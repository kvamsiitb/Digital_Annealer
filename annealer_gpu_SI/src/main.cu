#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <unistd.h>
#include <getopt.h>

#include <vector>
#include <chrono>

#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <algorithm>

#define THREADS 512 //or more threads gpu crashes
#define BREAK_UPDATE_VAL 2//1000 
#define TCRIT 2.26918531421f

#include "annealer_gpu_SI/utils.hpp"

#define PERCENTAGE_CHANGE_MAX_ENERGY 0.01f
#define BREAK_AFTER_ITERATION 0.05f
//__constant__ float kd_floats[1000000];
void printVecOfVec(std::vector<float> adjMat)
{
	std::cout << "\n";
	for (int j = 0; j < sqrt(adjMat.size()); j++) {
		for (int i = 0; i < sqrt(adjMat.size()); i++)
		{
			std::cout << adjMat[i + sqrt(adjMat.size())*j] << '\t';
		}
		std::cout << "\n";
	}

}

// float atomicMin
__device__ __forceinline__ float mAtomicMin(float *address, float val)
{
	int ret = __float_as_int(*address);
	while (val < __int_as_float(ret))
	{
		int old = ret;
		if ((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
			break;
	}
	return __int_as_float(ret);
}

//
__device__ __forceinline__ float mAtomicMax(float *address, float val)
{
	int ret = __float_as_int(*address);
	while (val > __int_as_float(ret))
	{
		int old = ret;
		if ((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
			break;
	}
	return __int_as_float(ret);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void d_debug_kernel(float* gpuAdjMat, unsigned int* gpuAdjMatSize, signed char* gpu_spins, signed char* gpu_spins_1, const unsigned int* gpu_num_spins);


__global__ void init_best_energy(float* total_energy, float* best_energy, bool init = false)
{
	if (init)
	{
		best_energy[0] = total_energy[0];
		printf("initial energy %.6f \n", total_energy[0]);
	}
	else
	{
		mAtomicMin(best_energy, total_energy[0]);
		printf(" best_energy, total_energy %.6f %.6f \n", best_energy[0], total_energy[0]);
	}
}



// Initialize lattice spins
__global__ void init_spins_total_energy(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	const float* __restrict__ randvals,
	signed char* gpuSpins,
	signed char* gpuSpins_1,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	float* total_energy,
	hiprandState * state,
	unsigned long seed);

// fINAL lattice spins
__global__ void final_spins_total_energy(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	signed char* gpuSpins,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	float* total_energy);

__global__ void alter_spin(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	const float* __restrict__ randvals,
	signed char* gpuLatSpin,
	signed char* gpuLatSpin_1,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	const float beta,
	float* total_energy,
	hiprandState* globalState,
	unsigned int* dev_select_spin_arr,
  clock_t *timer);


	__global__ void d_avg_magnetism(signed char* gpuSpins, const unsigned int* gpu_num_spins, float* avg_magnetism)	
{	
  unsigned int p_Id = threadIdx.x;	
  	
	__shared__ float sh_mem_spins_Energy[THREADS];	
  sh_mem_spins_Energy[p_Id] = 0;	
  __syncthreads();	
  	
  	int num_iter = (gpu_num_spins[0] + THREADS - 1) / THREADS;	
  	// num_iter data chucks 	
	for (int i = 0; i < gpu_num_spins[0]; i++)	
	{	
		// p_Id (worker group)	
		if (p_Id + i * THREADS < gpu_num_spins[0])	
		{	
   // @R  (- 1.f ) * gpuAdjMat * gpuSpins  // https://editor.mergely.com/	
			sh_mem_spins_Energy[p_Id] += ((float)gpuSpins[p_Id + i * THREADS]); 	
		}	
	}	
	__syncthreads();	
 	
   for (int off = blockDim.x/2; off; off /= 2) {	
     if (threadIdx.x < off) {	
         sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];	
       }	
   __syncthreads();	
   }	
   	
	if (p_Id == 0)	
	{	
      avg_magnetism[0] = sh_mem_spins_Energy[0]/gpu_num_spins[0];	
    // printf("$$$$$$$$$$$$$$$$$ Average magnetism %.6f \n",  avg_magnetism[0]);	
  }	
}

// Initialize lattice spins
__global__ void preprocess_max_cut_from_ising(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	signed char* gpuSpins,
	const unsigned int* gpu_num_spins,
	float* max_cut_value,
	int* plus_one_spin,
	int* minus_one_spin);

std::vector<double> create_beta_schedule_linear(uint32_t num_sweeps, double beta_start, double beta_end = -1.f);

static void usage(const char *pname) {

	const char *bname = nullptr;//@R = rindex(pname, '/');

	fprintf(stdout,
		"Usage: %s [options]\n"
		"options:\n"
		"\t-i|--J_Matrix_file <FILENAME>\n"
		"\t\tConnectivity matrix (no multiple connection between same nodes)\n"
		"\n"
		"\t-x|--start temperature <FLOAT>\n"
		"\t\t \n"
		"\n"
		"\t-y|--stop temperature <FLOAT>\n"
		"\t\tnumber of lattice columns\n"
		"\n"
		"\t-n|--niters <INT>\n"
		"\t\tnumber of iterations\n"
		"\n"
		"\t-n|--sweeps_per_beta <INT>\n"
		"\t\tnumber of sweep per temperature\n"
		"\n"
		"\t-s|--seed <SEED>\n"
		"\t\tfix the starting point\n"
		"\n"
		"\t-s|--debug \n"
		"\t\t Print the final lattice value and shows avg magnetization at every temperature\n"
		"\n"
		"\t-o|--write-lattice\n"
		"\t\twrite final lattice configuration to file\n\n",
		bname);
	exit(EXIT_SUCCESS);
}

int main(int argc, char* argv[])
{

  std::string filename = "";//argv[1]
  float start_temp = 20.f;
  float stop_temp = 0.001f;
	unsigned long long seed = ((getpid()* rand()) & 0x7FFFFFFFF); //((GetCurrentProcessId()* rand()) & 0x7FFFFFFFF);
  
  unsigned int num_temps = 1000; //atoi(argv[2]);
	unsigned int num_sweeps_per_beta = 1;//atoi(argv[3]);
	
 
  bool write = false;
  bool debug = false;
 	std::cout << "Start parsing the file " << std::endl;

  while (1) {
		static struct option long_options[] = {
      {     "J_Matrix_file", required_argument, 0, 'a'},
			{     "start_temp", required_argument, 0, 'x'},
			{     "stop_temp", required_argument, 0, 'y'},
			{          "seed", required_argument, 0, 's'},
			{        "niters", required_argument, 0, 'n'},
			{ "sweeps_per_beta", required_argument, 0, 'm'},
			{ "write-lattice",       no_argument, 0, 'o'},
      {          "debug",       no_argument, 0, 'd'},
			{          "help",       no_argument, 0, 'h'},
			{               0,                 0, 0,   0}
		};

		int option_index = 0;
		int ch = getopt_long(argc, argv, "a:x:y:s:n:m:odh", long_options, &option_index);
		if (ch == -1) break;

		switch (ch) {
		case 0:
			break;
   case 'a':
			filename = (optarg); break;
		case 'x':
			start_temp = atof(optarg); break;
		case 'y':
			stop_temp = atof(optarg); break;
		case 's':
			seed = atoll(optarg);
			break;
		case 'n':
			num_temps = atoi(optarg); break;
		case 'm':
			num_sweeps_per_beta = atoi(optarg); break;
		case 'o':
			write = true; break;
 		case 'd':
			debug = true; break;
		case 'h':
			usage(argv[0]); break;
		case '?':
			exit(EXIT_FAILURE);
		default:
			fprintf(stderr, "unknown option: %c\n", ch);
			exit(EXIT_FAILURE);
		}
	}

std::cout << "filename " << filename << " start temp " << start_temp << " stop temp " << stop_temp << " seed " << seed << " num temp " << num_temps << " num sweeps " <<  num_sweeps_per_beta << std::endl;
	std::vector<float> adjMat;// float
 	double starttime = rtclock();
	ParseData parseData(filename, adjMat);
	
	double endtime = rtclock();
  
  if(debug)
  	printtime("ParseData time: ", starttime, endtime);

	unsigned int adj_mat_size = adjMat.size();
	auto graphs_data = parseData.getDataDims();//sqrt(adjMat.size());
	unsigned int num_spins = graphs_data.at(0);
	unsigned int CPU_THREADS = THREADS;//(num_spins < 32) ? num_spins : 32; 
//	hipMemcpyToSymbol(HIP_SYMBOL( &THREADS), &CPU_THREADS, sizeof(unsigned int));
	// Setup cuRAND generator
	
  std::cout << "adj_mat_size: " << adj_mat_size << " num_spins: " << num_spins << " num of temperature "<< num_temps << " num_sweeps per beta "<< num_sweeps_per_beta << std::endl;
	hiprandGenerator_t rng;
	
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	hiprandSetPseudoRandomGeneratorSeed(rng, seed);
	float *gpu_randvals;// same as spins
	gpuErrchk(hipMalloc((void**)&gpu_randvals, (num_spins) * sizeof(float)));


	float *gpuAdjMat;
	gpuErrchk(hipMalloc((void**)&gpuAdjMat, (adj_mat_size) * sizeof(float)));
 
	starttime = rtclock();
	gpuErrchk(hipMemcpy(gpuAdjMat, adjMat.data(), (adj_mat_size) * sizeof(float), hipMemcpyHostToDevice));
  endtime = rtclock();
   
  if(debug)
     printtime("J Matrix data transfer time: ", starttime, endtime);
	//printVecOfVec(adjMat);

	unsigned int* gpu_adj_mat_size;
	gpuErrchk(hipMalloc((void**)&gpu_adj_mat_size, sizeof(*gpu_adj_mat_size)));
	gpuErrchk(hipMemcpy(gpu_adj_mat_size, &adj_mat_size, sizeof(*gpu_adj_mat_size), hipMemcpyHostToDevice));

	unsigned int* gpu_num_spins;
	gpuErrchk(hipMalloc((void**)&gpu_num_spins, sizeof(*gpu_num_spins)));
	gpuErrchk(hipMemcpy(gpu_num_spins, &num_spins, sizeof(*gpu_num_spins), hipMemcpyHostToDevice));
	adjMat.clear();// deallcoate vector //@ERROR

	int* gpu_plus_one_spin;
	hipHostAlloc(&gpu_plus_one_spin, sizeof(int), 0);

	int* gpu_minus_one_spin;
	hipHostAlloc(&gpu_minus_one_spin, sizeof(int), 0);

	int* gpu_best_plus_one_spin;
	hipHostAlloc(&gpu_best_plus_one_spin, sizeof(int), 0);
	gpu_best_plus_one_spin[0] = 0;

	int* gpu_best_minus_one_spin;
	hipHostAlloc(&gpu_best_minus_one_spin, sizeof(int), 0);
	gpu_best_minus_one_spin[0] = 0;

	unsigned int* gpu_select_spin_arr;
	hipHostAlloc(&gpu_select_spin_arr, sizeof(*gpu_select_spin_arr), 0);
	gpu_select_spin_arr[0] = 0;


	
	float* gpu_total_energy;
	hipHostAlloc(&gpu_total_energy, sizeof(float), 0);

	float* gpu_best_energy;
	hipHostAlloc(&gpu_best_energy, sizeof(float), 0);

	float* gpu_max_cut_value;
	hipHostAlloc(&gpu_max_cut_value, sizeof(float), 0);

	float* gpu_best_max_cut_value;
	hipHostAlloc(&gpu_best_max_cut_value, sizeof(float), 0);
	gpu_best_max_cut_value[0] = -1000.f;

	float* gpu_avg_magnetism;	
	hipHostAlloc(&gpu_avg_magnetism, sizeof(*gpu_avg_magnetism), 0);	
	gpu_avg_magnetism[0] = 0.f;
 
	// Setup spin values
	signed char *gpu_spins;
	gpuErrchk(hipMalloc((void**)&gpu_spins, num_spins * sizeof(*gpu_spins)));

	// Setup spin values
	signed char *gpu_spins_1;
	gpuErrchk(hipMalloc((void**)&gpu_spins_1, num_spins * sizeof(*gpu_spins_1)));

	// Setup spin values
	float *gpu_hamiltonian_per_spin;
	gpuErrchk(hipMalloc((void**)&gpu_hamiltonian_per_spin, num_spins * sizeof(float)));

	std::cout << "initialize spin values " << std::endl;
	//int blocks = (num_spins + THREADS - 1) / THREADS;
	hiprandGenerateUniform(rng, gpu_randvals, num_spins);
	
  //d_debug_kernel<<< 1, 1>>>(gpuAdjMat, gpu_adj_mat_size, gpu_num_spins);
  
  /*********************Time Analysis***********************************/
  clock_t *dtimer = nullptr;
  clock_t timer[num_spins * 2];
  gpuErrchk(hipMalloc((void **)&dtimer, sizeof(clock_t) * num_spins * 2));
  /*********************End Time Analysis***********************************/
// is a seed for random number generator
	time_t t;
	time(&t);
 
	//  create random states    
	hiprandState* devRanStates;
	hipMalloc(&devRanStates, num_spins * sizeof(hiprandState));
 	
   starttime = rtclock();

	init_spins_total_energy << < num_spins, THREADS >> > (gpuAdjMat, gpu_adj_mat_size,
		gpu_randvals,
		gpu_spins,
		gpu_spins_1,
		gpu_num_spins,
		gpu_hamiltonian_per_spin,
		gpu_total_energy,
		devRanStates,
		(unsigned long)t);
  
  hipDeviceSynchronize();
      
 	 endtime = rtclock();

	printtime("init_spins values and calculate total Energy time: ", starttime, endtime);
 

	gpuErrchk(hipPeekAtLastError());

	gpu_best_energy[0] = gpu_total_energy[0];

	std::cout << "start annealing with initial energy: " << gpu_best_energy[0] << std::endl;
	std::vector<double> beta_schedule = create_beta_schedule_linear(num_temps, start_temp, stop_temp);


  std::string out_filename = "avgmagnet_";  
  std::string in_adjmat = filename;
  {
    // Find position of '_' using find()
    int pos = in_adjmat.find_last_of("_");
    // Copy substring after pos
    std::string sub = in_adjmat.substr(pos + 1);
    out_filename += sub;
  }

 	FILE* fptr = fopen(out_filename.c_str() , "w");

	auto t0 = std::chrono::high_resolution_clock::now();
 
// temperature 
	for (int i = 0; i < beta_schedule.size(); i++)
	{
	 int no_update = 0;
	 hipEvent_t start, stop;
   if(debug)
   {   
     // @ Debugging
     
     hipEventCreate(&start);
     hipEventCreate(&stop);
    }         
         
      for(int ii = 0; ii < num_sweeps_per_beta; ii++)
	    {   
        //int prev_energy = gpu_total_energy[0];
 
        hiprandGenerateUniform(rng, gpu_randvals, num_spins);
   if(debug)
   {         
        hipEventRecord(start); 
   }
      	alter_spin << < num_spins, THREADS >> > (gpuAdjMat, gpu_adj_mat_size,
      			gpu_randvals,
      			gpu_spins,
      			gpu_spins_1,
      			gpu_num_spins,
      			gpu_hamiltonian_per_spin,
      			beta_schedule.at(i),
      			gpu_total_energy,
      			devRanStates,
      			gpu_select_spin_arr,
            dtimer);
    if(debug)
    {
       hipEventRecord(stop);   
         hipEventSynchronize(stop);
         float milliseconds = 0;
         hipEventElapsedTime(&milliseconds, start, stop);
         printf("Elapse time : %f ms \n", milliseconds);
    }     
       hipDeviceSynchronize();
       
       if(gpu_total_energy[0] > gpu_best_energy[0])
           no_update = 0;
       
       gpu_best_energy[0] = std::min(gpu_total_energy[0], gpu_best_energy[0]);
    	 
       if (  (gpu_best_energy[0] - gpu_total_energy[0]) < (PERCENTAGE_CHANGE_MAX_ENERGY) * gpu_best_energy[0])
  		  	no_update = 0;
  		 else
  		  	no_update++;
  	//	printf("cur engy %.1f best engy %.1f \n", gpu_total_energy[0], gpu_best_energy[0]);
  		if (no_update > (BREAK_AFTER_ITERATION) * num_sweeps_per_beta)
  			{
        break;
        }

              
// @R Debugging
if(debug)
{
   if(gpu_select_spin_arr[0]%2 == 0)	
   {	
       d_avg_magnetism << < 1, THREADS >> >(gpu_spins_1, gpu_num_spins, gpu_avg_magnetism);	
   }	
   else	
   {	
       d_avg_magnetism << < 1, THREADS >> >(gpu_spins, gpu_num_spins, gpu_avg_magnetism);   	
   }
}     	
           hipDeviceSynchronize();      
           gpu_select_spin_arr[0]++;
	
		 gpuErrchk(hipPeekAtLastError());         		 
 	  }
          
   
  /*********************Time Analysis***********************************/
  gpuErrchk(hipMemcpy(timer, dtimer, sizeof(clock_t) * num_spins * 2, hipMemcpyDeviceToHost));
  if(debug)
    fprintf(fptr, "Temperature %.6f magnet %.6f \n", 1.f/beta_schedule.at(i),  gpu_avg_magnetism[0]); 
  clock_t minStart = timer[0];
  clock_t maxEnd = timer[num_spins];
/* // To find total time taken to launch the kernel
    for (int i = 1; i < num_spins; i++)
    {
        minStart = timer[i] < minStart ? timer[i] : minStart;
        maxEnd = timer[num_spins+i] > maxEnd ? timer[num_spins+i] : maxEnd;
    }

 */
 //   printf("Total clocks = %Lf\n", (long double)(maxEnd - minStart));

  /*********************End Time Analysis***********************************/
	}
  hipFree(dtimer);
 
	auto t1 = std::chrono::high_resolution_clock::now();

	double duration = (double)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();

  fprintf(fptr, "duration %.3f \n", (duration * 1e-6) );
  fclose(fptr);

 
 // @R Debugging 
/*	d_debug_kernel << < 1, 1 >> > (gpuAdjMat,
		gpu_adj_mat_size,
		gpu_spins,
		gpu_spins_1,
		gpu_num_spins);
*/   


  
  signed char cpu_spins[num_spins];

	gpu_max_cut_value[0] = 0.f;
	gpu_plus_one_spin[0] = 0;
	gpu_minus_one_spin[0] = 0;
  gpu_total_energy[0] = 0;
   if(gpu_select_spin_arr[0]%2 == 0)	
   {
    
	  final_spins_total_energy << < num_spins, THREADS >> > (gpuAdjMat, gpu_adj_mat_size,
			 gpu_spins_1,
			 gpu_num_spins,
			 gpu_hamiltonian_per_spin,
			 gpu_total_energy); 
        
    preprocess_max_cut_from_ising << < num_spins, THREADS >> > (gpuAdjMat,
  				gpu_adj_mat_size,
  				gpu_spins_1,
  				gpu_num_spins,
  				gpu_max_cut_value,
  				gpu_plus_one_spin,
  				gpu_minus_one_spin);
  
		 hipDeviceSynchronize();
     printf("@@@@ cur engy %.1f best engy %.1f \n", gpu_total_energy[0], gpu_best_energy[0]);
     gpuErrchk(hipMemcpy(cpu_spins, gpu_spins_1, num_spins * sizeof(*gpu_spins_1), hipMemcpyDeviceToHost));
    }    
   else	
   {
   
   		 final_spins_total_energy << < num_spins, THREADS >> > (gpuAdjMat, gpu_adj_mat_size,
  			 gpu_spins,
  			 gpu_num_spins,
  			 gpu_hamiltonian_per_spin,
  			 gpu_total_energy);
          
       preprocess_max_cut_from_ising << < num_spins, THREADS >> > (gpuAdjMat,
  				gpu_adj_mat_size,
  				gpu_spins,
  				gpu_num_spins,
  				gpu_max_cut_value,
  				gpu_plus_one_spin,
  				gpu_minus_one_spin);
  
  		 hipDeviceSynchronize();
       printf("@@@@ cur engy %.1f best engy %.1f \n", gpu_total_energy[0], gpu_best_energy[0]);          
       gpuErrchk(hipMemcpy(cpu_spins, gpu_spins_1, num_spins * sizeof(*gpu_spins), hipMemcpyDeviceToHost));
   }     
        
			gpu_best_max_cut_value[0] = std::max(gpu_best_max_cut_value[0], gpu_max_cut_value[0]);
			gpu_best_plus_one_spin[0] = std::max(gpu_best_plus_one_spin[0], gpu_plus_one_spin[0]);
			gpu_best_minus_one_spin[0] = std::max(gpu_best_minus_one_spin[0], gpu_minus_one_spin[0]);
			//		gpu_low_minus_one_spin_check[0] = std::min(gpu_low_minus_one_spin_check[0], gpu_plus_one_spin[0]);
			printf("cur engy %.1f best engy %.1f curr cut %.1f best cut %.1f with best +1 %d and -1 %d \n", gpu_total_energy[0], gpu_best_energy[0], gpu_max_cut_value[0], gpu_best_max_cut_value[0], gpu_best_plus_one_spin[0], gpu_best_minus_one_spin[0]);

 if(debug)
 {

  std::string spins_filename = "spins_";  
  
  std::string adjmat = filename;

  {
    // Find position of '_' using find()
    int pos = adjmat.find_last_of("_");
    // Copy substring after pos
    std::string sub = adjmat.substr(pos + 1);
    spins_filename += sub;
  }

 	FILE* fptr1 = fopen(spins_filename.c_str() , "w");
  for(int i = 0; i < num_spins; i++)
  {
        fprintf(fptr1, "%d\t",  (int)cpu_spins[i]);

  }  
  fprintf(fptr1,"\n\n\n");
  // fprintf(fptr1,"\tbest energy value: %.6f\n", gpu_best_energy[0] );
  fprintf(fptr1,"\ttotal energy value: %.6f\n", gpu_total_energy[0] );
  fprintf(fptr1,"\tbest max cut value: %.6f\n", gpu_best_max_cut_value[0]);
	fprintf(fptr1," \telapsed time in sec: %.6f\n", duration * 1e-6 );
  fclose(fptr1);
  
 }
	std::cout << "\tbest energy value: " << gpu_best_energy[0] << std::endl;
	std::cout << "\tbest max cut value: " << gpu_best_max_cut_value[0] << std::endl;
	std::cout << "\telapsed time in sec: " << duration * 1e-6 << std::endl;
 
	hipFree(gpu_randvals);
	hipFree(gpuAdjMat);
	hipFree(gpu_adj_mat_size);
	hipFree(gpu_num_spins);
	//hipFree(gpu_total_energy);
	//hipFree(gpu_best_energy);
	hipFree(gpu_hamiltonian_per_spin);
	hipFree(gpu_spins);
	hipFree(gpu_spins_1);
	//hipFree(gpu_beta);
	return 0;
}




#define CORRECT 1

#if CORRECT
__global__ void alter_spin(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	const float* __restrict__ randvals,
	signed char* gpuLatSpin,
	signed char* gpuLatSpin_1,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	const float beta,
	float* total_energy,
	hiprandState* globalState,
	unsigned int* dev_select_spin_arr,
  clock_t *timer) {

	unsigned int vertice_Id = blockIdx.x;
	unsigned int p_Id = threadIdx.x;    //32 worker threads 
	// for each neighour of vertex id pull the gpucurrentupdate[i] and place it in the shared memory

	// shared  spin_v0|spin_v1|.......|J_spin0| J_spin1| J_spin2|..
	__shared__ float sh_mem_spins_Energy[THREADS];
  sh_mem_spins_Energy[p_Id] = 0;
  __syncthreads();

	float current_spin_shared_mem;

	if (dev_select_spin_arr[0] % 2 == 0)
		current_spin_shared_mem = (float)gpuLatSpin[vertice_Id];
	else
		current_spin_shared_mem = (float)gpuLatSpin_1[vertice_Id];


	unsigned int stride_jump_each_vertice = sqrt((float)gpuAdjMatSize[0]);
	unsigned int num_spins = gpu_num_spins[0];
	int num_iter = int((num_spins) / THREADS) + 1;// @R (num_spins + THREADS - 1) / THREADS;

	// placing all the spins data into the shared memory..
	// hence, decouple the spins to the global spins

 
	for (int i = 0; i < num_iter; i++)
	{
		if (p_Id + i * THREADS < num_spins)
		{
			float current_spin;
			if (dev_select_spin_arr[0] % 2 == 0)
				current_spin = (float)gpuLatSpin[p_Id + i * THREADS];
			else
				current_spin = (float)gpuLatSpin_1[p_Id + i * THREADS];
        
			sh_mem_spins_Energy[p_Id] += gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * (current_spin);
 		       
		}
	}
	__syncthreads();


  for (int off = blockDim.x/2; off; off /= 2) {
     if (threadIdx.x < off) {
         sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];
       }
   __syncthreads();
   }
   
  __syncthreads();
	
	if (p_Id == 0)
	{
  
	  float vertice_change_energy = - 1.f * sh_mem_spins_Energy[0];
	   
    float change_in_energy = - 2.f * vertice_change_energy * current_spin_shared_mem; // final energy - current energy

    if(change_in_energy > 0)
    {
      		float acceptance_ratio = exp( 2.f * beta * vertice_change_energy * current_spin_shared_mem);
      		if (randvals[vertice_Id] < acceptance_ratio) // low temp
      		{  
      			if (dev_select_spin_arr[0] % 2 == 0)
      				gpuLatSpin_1[vertice_Id] = (signed char)(1.f * current_spin_shared_mem);
      			else
      				gpuLatSpin[vertice_Id] = (signed char)(1.f * current_spin_shared_mem); 
      		}      
      	else { 
            			if (dev_select_spin_arr[0] % 2 == 0)
            				gpuLatSpin_1[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
            			else
            				gpuLatSpin[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
    			        __threadfence();
                  atomicAdd(total_energy, (change_in_energy) );
             } 
    } 
   	else {
    
       		float acceptance_ratio = exp( -2.f * beta * vertice_change_energy * current_spin_shared_mem);
      		if (randvals[vertice_Id] < acceptance_ratio)// change is good and low temp
      		{   
      			if (dev_select_spin_arr[0] % 2 == 0)
      				gpuLatSpin_1[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
      			else
      				gpuLatSpin[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
      			__threadfence();    
      			atomicAdd(total_energy, (change_in_energy) );
      		}      
            	else {
          			if (dev_select_spin_arr[0] % 2 == 0)
          				gpuLatSpin_1[vertice_Id] = (signed char)( current_spin_shared_mem);
          			else
          				gpuLatSpin[vertice_Id] = (signed char)( current_spin_shared_mem);
                }
		 } 
	}

  /*********************timing ******************************/
  //if (p_Id == 0) timer[vertice_Id+gridDim.x] = clock();
  /*********************End timing ******************************/

}


#endif

// Initialize lattice spins
__global__ void init_spins_total_energy(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	const float* __restrict__ randvals,
	signed char* gpuSpins,
	signed char* gpuSpins_1,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	float* total_energy,
	hiprandState * state,
	unsigned long seed) {

	unsigned int vertice_Id = blockIdx.x; // actual spin id in this threadBlock
	unsigned int p_Id = threadIdx.x;// which worker id

	if (p_Id == 0)
	{
		float randval = randvals[vertice_Id];
		signed char val = (randval < 0.5f) ? -1 : 1;
		gpuSpins[vertice_Id] = val;// random spin init.
		gpuSpins_1[vertice_Id] = val;// random spin init.
		hiprand_init(seed, blockIdx.x, 0, &state[blockIdx.x]);
	}
	__syncthreads();

	__shared__ float sh_mem_spins_Energy[THREADS];
  sh_mem_spins_Energy[p_Id] = 0;
  __syncthreads();
  
	unsigned int stride_jump_each_vertice = sqrt((float)gpuAdjMatSize[0]);
	unsigned int num_spins = gpu_num_spins[0];
	int num_iter = (num_spins + THREADS - 1) / THREADS;

	// num_iter data chucks 
	for (int i = 0; i < num_iter; i++)
	{
		// p_Id (worker group)
		if (p_Id + i * THREADS < num_spins)
		{
   // @R  (- 1.f ) * gpuAdjMat * gpuSpins  // https://editor.mergely.com/
			sh_mem_spins_Energy[p_Id] += (- 1.f ) * gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * ((float)gpuSpins[p_Id + i * THREADS]); 
		}
	}
	__syncthreads();


  for (int off = blockDim.x/2; off; off /= 2) {
     if (threadIdx.x < off) {
         sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];
       }
   __syncthreads();
   }
 
  __syncthreads(); 
	if (p_Id == 0)
	{
 
		float vertice_energy = ((float)gpuSpins[vertice_Id]) * sh_mem_spins_Energy[0];
		hamiltonian_per_spin[vertice_Id] = vertice_energy;// each threadblock updates its own memory location

//		printf("vertice_energy  %f \n", vertice_energy);
		atomicAdd(total_energy, vertice_energy);
	}

	//        printf("%d total %.1f",blockIdx.x, total_energy);
}

// fINAL lattice spins
__global__ void final_spins_total_energy(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	signed char* gpuSpins,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	float* total_energy) {

	unsigned int vertice_Id = blockIdx.x; // actual spin id in this threadBlock
	unsigned int p_Id = threadIdx.x;// which worker id

	__shared__ float sh_mem_spins_Energy[THREADS];
	sh_mem_spins_Energy[p_Id] = 0;
	__syncthreads();

	unsigned int stride_jump_each_vertice = sqrt((float)gpuAdjMatSize[0]);
	unsigned int num_spins = gpu_num_spins[0];
	int num_iter = (num_spins + THREADS - 1) / THREADS;

	// num_iter data chucks 
	for (int i = 0; i < num_iter; i++)
	{
		// p_Id (worker group)
		if (p_Id + i * THREADS < num_spins)
		{
			// @R  (- 1.f ) * gpuAdjMat * gpuSpins  // https://editor.mergely.com/
			sh_mem_spins_Energy[p_Id] += (-1.f) * gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * ((float)gpuSpins[p_Id + i * THREADS]);
		}
	}
	__syncthreads();


	for (int off = blockDim.x / 2; off; off /= 2) {
		if (threadIdx.x < off) {
			sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];
		}
		__syncthreads();
	}

  __syncthreads();
	if (p_Id == 0)
	{

		float vertice_energy = ((float)gpuSpins[vertice_Id]) * sh_mem_spins_Energy[0];
		hamiltonian_per_spin[vertice_Id] = vertice_energy;// each threadblock updates its own memory location

		//printf("vertice_energy  %d %f \n",vertice_Id, vertice_energy);
		atomicAdd(total_energy, vertice_energy);
	}

	//        printf("%d total %.1f",blockIdx.x, total_energy);
}

// Initialize lattice spins
__global__ void preprocess_max_cut_from_ising(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	signed char* gpuSpins,
	const unsigned int* gpu_num_spins,
	float* max_cut_value,
	int* plus_one_spin,
	int* minus_one_spin) {

	unsigned int vertice_Id = blockIdx.x; // actual spin id in this threadBlock
	unsigned int p_Id = threadIdx.x;// which worker id
	float current_spin_row = (float)gpuSpins[vertice_Id];

	__shared__ float sh_mem_spins_Energy[THREADS];
  sh_mem_spins_Energy[p_Id] = 0;
  __syncthreads();

	unsigned int stride_jump_each_vertice = sqrt((float)gpuAdjMatSize[0]);
	unsigned int num_spins = gpu_num_spins[0];
	int num_iter = (num_spins + THREADS - 1) / THREADS;

	// num_iter data chucks 
	for (int i = 0; i < num_iter; i++)
	{
		// p_Id (worker group)
		if (p_Id + i * THREADS < num_spins)
		{
			sh_mem_spins_Energy[p_Id] += gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * (1.f - (current_spin_row * (float)gpuSpins[p_Id + i * THREADS]));
		}
	}
	__syncthreads();

  for (int off = blockDim.x/2; off; off /= 2) {
     if (threadIdx.x < off) {
         sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];
       }
   __syncthreads();
   }
   
	if (p_Id == 0)
	{

		float vertice_energy;
/*
		for (int i = 0; i < THREADS; i++)
		{
			vertice_energy += sh_mem_spins_Energy[i];
		}
*/   
		vertice_energy = (0.5f) * sh_mem_spins_Energy[0];

		atomicAdd(max_cut_value, vertice_energy);

		if (current_spin_row == 1.f)
			atomicAdd(plus_one_spin, 1);
		else
			atomicAdd(minus_one_spin, 1);
	}

	//       
}


std::vector<double> create_beta_schedule_linear(uint32_t num_sweeps, double beta_start, double beta_end)
{
	std::vector<double> beta_schedule;
	double beta_max;
	if (beta_end == -1)
		beta_max = (1/1000)*beta_start;//  here temperature will be zero when beta_max is 1000.f
	else
		beta_max = beta_end;
	double diff = (beta_start - beta_max) / (num_sweeps - 1);// A.P 3.28 - 0.01 inverse value increa finnal decrease
	for (int i = 0; i < num_sweeps; i++)
	{
		double val = beta_start - (i)*diff;
		beta_schedule.push_back(( 1.f /val));
	}
	
	return beta_schedule;
}

__global__ void d_debug_kernel(float* gpuAdjMat, unsigned int* gpuAdjMatSize, signed char* gpu_spins, signed char* gpu_spins_1, const unsigned int* gpu_num_spins)
{
	/*		printf("Number of elements %d, number of spins %d \n", gpuAdjMatSize[0], gpu_num_spins[0]);
			for(int i = 0; i < gpuAdjMatSize[0]; i++)
			{
			   if(i%gpu_num_spins[0] == 0)
				   printf("\n");
			   printf("%.01f \t", gpuAdjMat[i]);
			}
			printf("\n");
	*/
	int ones = 0;
	int ones_1 = 0;
	for (int i = 0; i < gpu_num_spins[0]; i++)
	{
		printf("%d %.1f ", i, (float)gpu_spins[i]);
		if ((float)gpu_spins[i] == 1.f)
			ones++;
		if ((float)gpu_spins_1[i] == -1.f)
			ones_1++;
	}

	printf("\n");
	printf("\n");
	printf("%d %d \n", ones, ones_1);
	int m_ones = 0;
	int m_ones_1 = 0;
	for (int i = 0; i < gpu_num_spins[0]; i++)
	{
		printf("%d %.1f ", i, (float)gpu_spins_1[i]);
		if ((float)gpu_spins[i] == 1.f)
			m_ones++;
		if ((float)gpu_spins_1[i] == -1.f)
			m_ones_1++;
	}
	printf("\n");
	printf("\n");
	printf("%d %d\n", m_ones, m_ones_1);
}



#define COPY 0

#if COPY
// Correct

// https://stackoverflow.com/questions/57902066/how-to-generate-unique-random-integers-with-hiprand
__global__ void alter_spin(float* gpuAdjMat, unsigned int* gpuAdjMatSize,
	const float* __restrict__ randvals,
	signed char* gpuLatSpin,
	signed char* gpuLatSpin_1,
	const unsigned int* gpu_num_spins,
	float* hamiltonian_per_spin,
	const float beta,
	float* total_energy,
	float* best_energy,
	hiprandState* globalState,
	unsigned int* dev_select_spin_arr,
  clock_t *timer) {

	unsigned int vertice_Id = blockIdx.x;
	unsigned int p_Id = threadIdx.x;    //32 worker threads 
	// for each neighour of vertex id pull the gpucurrentupdate[i] and place it in the shared memory

	// shared  spin_v0|spin_v1|.......|J_spin0| J_spin1| J_spin2|..
	__shared__ float sh_mem_spins_Energy[THREADS];
  sh_mem_spins_Energy[p_Id] = 0;
  __syncthreads();
/*	
	unsigned int rand;
	if(threadIdx.x == 0)
	{
		rand = hiprand(&globalState[blockIdx.x]);
		for(int i = 0; i < rand; i++)
		{
			;
		}

	   //printf("blockOrder %d,rand %u \n", blockIdx.x, rand);
	}
	__syncthreads();
*/

/*********************timing ******************************/
//if (p_Id == 0) timer[vertice_Id] = clock();
/*********************End timing ******************************/
	float current_spin_shared_mem;

	if (dev_select_spin_arr[0] % 2 == 0)
		current_spin_shared_mem = (float)gpuLatSpin[vertice_Id];
	else
		current_spin_shared_mem = (float)gpuLatSpin_1[vertice_Id];

 //current_spin_shared_mem = (float)gpuLatSpin[vertice_Id];

	unsigned int stride_jump_each_vertice = sqrt((float)gpuAdjMatSize[0]);
	unsigned int num_spins = gpu_num_spins[0];
	int num_iter = int((num_spins) / THREADS) + 1;// @R (num_spins + THREADS - 1) / THREADS;

	// placing all the spins data into the shared memory..
	// hence, decouple the spins to the global spins

 
	for (int i = 0; i < num_iter; i++)
	{
		if (p_Id + i * THREADS < num_spins)
		{
			float current_spin;
			if (dev_select_spin_arr[0] % 2 == 0)
				current_spin = (float)gpuLatSpin[p_Id + i * THREADS];
			else
				current_spin = (float)gpuLatSpin_1[p_Id + i * THREADS];
        
        // @R ( - 1.f ) * gpuAdjMat * current_spin
			sh_mem_spins_Energy[p_Id] += gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * (current_spin);
			//*	sh_mem_spins_Energy[p_Id + i * THREADS + num_spins] = ( - 1.f ) * gpuAdjMat[p_Id + (i * THREADS) + (vertice_Id * stride_jump_each_vertice)] * (current_spin); 		       
		}
	}
	__syncthreads();


  for (int off = blockDim.x/2; off; off /= 2) {
     if (threadIdx.x < off) {
         sh_mem_spins_Energy[threadIdx.x] += sh_mem_spins_Energy[threadIdx.x + off];
       }
   __syncthreads();
   }
   
  __syncthreads();
	// prefix sum to be implemented
	if (p_Id == 0)
	{
     // spins_A -- > | 1| -1 | given J_ij
     // spins_B -- > | -1| 1 |
     // local H(a_i)
	  float vertice_change_energy = 0.f;
	  vertice_change_energy =  sh_mem_spins_Energy[0];
		//*  float current_spin_shared_mem = sh_mem_spins_Energy[vertice_Id];
    // 	if( (2.f * vertice_change_energy * current_spin_shared_mem) > 0 ) //< hamiltonian_per_spin[vertice_Id] )
   //{
	  // updated the local value of vertex no id
   
  // printf("Print &&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&& %d %d %.2f, %.f, %.f \n", vertice_Id, dev_select_spin_arr[0], randvals[vertice_Id], sh_mem_spins_Energy[0], current_spin_shared_mem);
     
    float change_in_energy = - 2.f * vertice_change_energy * current_spin_shared_mem; // final energy - current energy
    // 2 example all the neighbouring spins +1 and center +1 or otherwise. it is positive
    if(change_in_energy < 0)
    {
      		float acceptance_ratio = exp(- 2.f * beta * vertice_change_energy * current_spin_shared_mem);
      		if (randvals[vertice_Id] < acceptance_ratio) // low temp
      		{
    
          // printf("what is this $ 1 temperature %.2f randval %.2f acceptance %.2f\n", 1.f/beta, randvals[vertice_Id], acceptance_ratio);
     
      			if (dev_select_spin_arr[0] % 2 == 0)
      				gpuLatSpin_1[vertice_Id] = (signed char)(1.f * current_spin_shared_mem);
      			else
      				gpuLatSpin[vertice_Id] = (signed char)(1.f * current_spin_shared_mem);
      			//printf("1, %d %f", vertice_Id, -current_spin_shared_mem);

 
      		}      
      	else { // let the bad configuration stay  // high temp
     
       // printf("what is this $$$$ 2 temperature %.2f randval %.2f acceptance %.2f\n", 1.f/beta, randvals[vertice_Id], acceptance_ratio);
            			if (dev_select_spin_arr[0] % 2 == 0)
            				gpuLatSpin_1[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
            			else
            				gpuLatSpin[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
            			//printf("3, %d %f", vertice_Id, current_spin_shared_mem);
    			        __threadfence();
                  atomicAdd(total_energy, change_in_energy);
             } 
    } 
   	else {
    
       		float acceptance_ratio = exp(2.f * beta * vertice_change_energy * current_spin_shared_mem);
      		if (randvals[vertice_Id] < acceptance_ratio)// change is good and low temp
      		{
       
        // printf("what is this $$$$$$$ 3 temperature %.2f randval %.2f acceptance %.2f\n", 1.f/beta, randvals[vertice_Id], acceptance_ratio);
     
      			if (dev_select_spin_arr[0] % 2 == 0)
      				gpuLatSpin_1[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
      			else
      				gpuLatSpin[vertice_Id] = (signed char)(-1.f * current_spin_shared_mem);
      			//printf("1, %d %f", vertice_Id, -current_spin_shared_mem);
      			__threadfence();    
      			atomicAdd(total_energy, change_in_energy);
      		}      
            	else {
             
          //     printf("what is this $$$$$$$$$$$$$$ 4 temperature %.2f randval %.2f acceptance %.2f\n", 1.f/beta, randvals[vertice_Id], acceptance_ratio);
          			if (dev_select_spin_arr[0] % 2 == 0)
          				gpuLatSpin_1[vertice_Id] = (signed char)( current_spin_shared_mem);
          			else
          				gpuLatSpin[vertice_Id] = (signed char)( current_spin_shared_mem);
                }
		 } 
  
	}

  /*********************timing ******************************/
  if (p_Id == 0) timer[vertice_Id+gridDim.x] = clock();
  /*********************End timing ******************************/

}


#endif
